#include<iostream>
#include "hip/hip_runtime.h"

__global__ void addone(int *a)
{
    *a = *a + 1;
    printf("add one \n");
}


int main()
{
    int a = 0;
    int *d_a;

    hipMalloc(&d_a, sizeof(int));
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);

    addone<<<1,32>>>(d_a);


    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);


    std::cout<<a<<std::endl;

    hipFree(d_a);

    return 0;
}